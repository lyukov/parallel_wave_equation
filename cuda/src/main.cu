#include "hip/hip_runtime.h"
#include <iostream>

using std::endl;

__global__ void sum_kernel(int *A, int *B, int *C) {
    //определить свой индекс int a = A[idx]; //считать нужный элемент A
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int b = B[idx]; // считать нужный элемент B
    C[idx] = a + b; //записать результат суммирования
}

int main(int argc, char **argv) {
    // Size of vectors
    int n = 100000;
    // Host vectors
    double *h_a, *h_b, *h_c;
    // Size, in bytes, of each vector
    size_t bytes = n * sizeof(double);
    // Allocate memory for each vector on host
    h_a = (double *) malloc(bytes);
    h_b = (double *) malloc(bytes);
    h_c = (double *) malloc(bytes);
    int i;
    // Initialize vectors on host
    for (i = 0; i < n; i++) {
        h_a[i] = sin(i) * sin(i);
        h_b[i] = cos(i) * cos(i);
    }

    // Device input vectors
    double *d_a, *d_b, *d_c;
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    // Copy host vectors to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    int blockSize = 1024;
    int gridSize = (n - 1) / blockSize + 1;
    // Execute the kernel
    sum_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    // Copy array back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    double maxError = 0;
    for (int i = 0; i < n; ++i) {
        double error = abs(h_c[i] - 1.0);
        maxError = error > maxError ? error : maxError;
    }
    std::cout << "Max error = " << maxError << endl;
}