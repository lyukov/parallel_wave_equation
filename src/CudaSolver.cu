#include "hip/hip_runtime.h"
#include "CudaSolver.cuh"
#include <cstdlib>

#define SAFE_CALL(CallInstruction) { \
    hipError_t cuerr = CallInstruction; \
    if(cuerr != hipSuccess) { \
        printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
            throw "error in CUDA API function, aborting..."; \
    } \
}

#define SAFE_KERNEL_CALL(KernelCallInstruction) { \
    KernelCallInstruction; \
    hipError_t cuerr = hipGetLastError(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
            throw "error in CUDA kernel launch, aborting..."; \
    } \
    cuerr = hipDeviceSynchronize(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
            throw "error in CUDA kernel execution, aborting..."; \
    } \
}

__constant__ int d_cfI;
__constant__ int d_cfJ;
__constant__ double d_h_x;
__constant__ double d_h_y;
__constant__ double d_h_z;
__constant__ double d_sqr_tau;

__device__
double &get_at(double *g, int i, int j, int k) {
    return g[i * d_cfI + j * d_cfJ + k];
}

__device__
double laplacian(double *g, int i, int j, int k) {
    double center = get_at(g, i, j, k);
    return (get_at(g, i - 1, j, k) - 2.0 * center + get_at(g, i + 1, j, k)) / (d_h_x * d_h_x) +
           (get_at(g, i, j - 1, k) - 2.0 * center + get_at(g, i, j + 1, k)) / (d_h_y * d_h_y) +
           (get_at(g, i, j, k - 1) - 2.0 * center + get_at(g, i, j, k + 1)) / (d_h_z * d_h_z);
}

__global__
void step(double *grid, double *previous_1, double *previous_2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
    get_at(grid, i, j, k) = 2.0 * get_at(previous_1, i, j, k) - get_at(previous_2, i, j, k) +
                            d_sqr_tau * laplacian(previous_1, i, j, k);
}

void makeStepWithCuda(Grid3D &grid, Grid3D &previous_1, Grid3D &previous_2,
                      double h_x, double h_y, double h_z, double sqr_tau) {
    dim3 blockSize = dim3(
            1,
            1,
            grid.shape[2] - 2
    );
    dim3 gridInBlocks = dim3(
            (grid.shape[0] - 2) / blockSize.x,
            (grid.shape[1] - 2) / blockSize.y,
            (grid.shape[2] - 2) / blockSize.z
    );

    size_t sizeInBytes = sizeof(double) * grid.size;

    double *d_grid;
    double *d_previous_1;
    double *d_previous_2;
    SAFE_CALL(hipMalloc((void **) &d_grid, sizeInBytes));
    SAFE_CALL(hipMalloc((void **) &d_previous_1, sizeInBytes));
    SAFE_CALL(hipMalloc((void **) &d_previous_2, sizeInBytes));
    SAFE_CALL(hipMemcpy(d_grid, grid.getFlatten().data(), sizeInBytes, hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(d_previous_1, previous_1.getFlatten().data(), sizeInBytes, hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(d_previous_2, previous_2.getFlatten().data(), sizeInBytes, hipMemcpyHostToDevice));

    hipMemcpyToSymbol(HIP_SYMBOL(d_cfI), &grid._cfI, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_cfJ), &grid._cfJ, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_h_x), &h_x, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(d_h_y), &h_y, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(d_h_z), &h_z, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(d_sqr_tau), &sqr_tau, sizeof(double));

    step<<<gridInBlocks, blockSize>>>(d_grid, d_previous_1, d_previous_2);

    SAFE_CALL(hipMemcpy(grid.getFlatten().data(), d_grid, sizeInBytes, hipMemcpyDeviceToHost));
}